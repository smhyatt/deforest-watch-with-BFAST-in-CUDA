#include "hip/hip_runtime.h"
#include <sys/time.h>
#include <string.h>
#include <stdlib.h>
#include <stdio.h>
#include <float.h>
#include <math.h>
#include <time.h>

#include "helper.cu.h"
#include "kernels-optim.cu.h"
#include "pbbKernels.cu.h"
#include "sequential.cu.h"

#define F32_MIN -FLT_MAX
typedef unsigned int uint;

/////////////////////////////////////////////////////////////////////////
///////////////////////// Naive kernels /////////////////////////////////
/////////////////////////////////////////////////////////////////////////


/////////////////////////////////////////////////////////////////////////
/////////////////////////////////////////////////////////////////////////
//// Helpers
/////////////////////////////////////////////////////////////////////////
/////////////////////////////////////////////////////////////////////////

int timeval_subtract(struct timeval *result, struct timeval *t2, struct timeval *t1) {
    unsigned int resolution=1000000;
    long int diff = (t2->tv_usec + resolution * t2->tv_sec) - (t1->tv_usec + resolution * t1->tv_sec);
    result->tv_sec = diff / resolution;
    result->tv_usec = diff % resolution;
    return (diff<0);
}

int gpuAssert(hipError_t code) {
    if(code != hipSuccess) {
      printf("GPU Error: %s\n", hipGetErrorString(code));
      return -1;
    }
    return 0;
}

//
// Cosmin's Matrix Transpose Wrapper from Weekly 3
//
void transposeTiled ( float*     inp_d,
                      float*     out_d,
                      const uint height,
                      const uint width,
                      const uint T
) {
   // 1. setup block and grid parameters
   unsigned int sh_mem_size = T * (T+1) * sizeof(float);
   int  dimy = (height+T-1) / T;
   int  dimx = (width +T-1) / T;
   dim3 block(T, T, 1);
   dim3 grid (dimx, dimy, 1);

   //2. execute the kernel
   matTransposeTiledKer<<< grid, block, sh_mem_size >>> (inp_d, out_d, height, width, T);
   // hipDeviceSynchronize();
}

void readNum(FILE *fp, char* buff) {
  int i = 0;
  char c = getc(fp);
  while( c != ',' && c != EOF) {
    buff[i++] = c;
    c = getc(fp);
  }
  buff[i] = '\0';
}

///////////////////////////////////////////////////////////////////////////////
///////////////////////////////////////////////////////////////////////////////
//// PROGRAM MAIN
///////////////////////////////////////////////////////////////////////////////
///////////////////////////////////////////////////////////////////////////////

int main(int argc, char const *argv[]) {
   if (argc != 2) {
      printf("Please include the name of the dataset.\n");
         return -1;
   }

///////////////////////////////////////////////////////////////////////////////
//// PARSING
///////////////////////////////////////////////////////////////////////////////

    FILE *fp, *fpim, *fpV;// *fpTEST;
    uint m = 0;

    // opening files with input and fpV for validating the results
    if (argv[1][0] == '2') {
        if (argv[1][1] == 's') {
            fp   = fopen("../data/sahara2C.in", "r");
            fpim = fopen("../data/sahara2Cimages.in", "r");
            fpV  = fopen("../data/sahara2val.data","a+");
            m = 2;
        } else {
            fp   = fopen("../data/peru2C.in", "r");
            fpim = fopen("../data/peru2Cimages.in", "r");
            fpV  = fopen("../data/peru2val.data","a+");
            m = 2;
        }
    } else {
        if (argv[1][0] == 's') {
            fp   = fopen("../data/saharaC.in", "r");
            fpim = fopen("../data/saharaCimages.in", "r");
            fpV  = fopen("../data/saharaval.data","a+");
        } else {
            fp   = fopen("../data/peruC.in", "r");
            fpim = fopen("../data/peruCimages.in", "r");
            fpV  = fopen("../data/peruval.data","a+");
        }
    }

    if (fp == NULL || fpim == NULL) {
        printf("Files not read.\n");
        return -1;
    }

    char input1[10], input2[10], input3[30], input4[30];
    char input5[30], input6[30], input7[50], input8[30];
    fscanf(fp, " %[^\n]  %[^\n]  %[^\n]  %[^\n] ", input1,input2,input3,input4);
    fscanf(fp, " %[^\n]  %[^\n]  %[^\n]  %[^\n] ", input5,input6,input7,input8);
    int  k    = atoi(input2);
    uint n    = (uint)atoi(input3);
    uint N    = (uint)atoi(input8);
    if (m == 0) {
        m = (uint)atoi(input7);
    }
    printf("\n\n%u\n\n", m);
    printf("\n\n%u\n\n", N);
    int   trend = atoi(input1);
    float freq  = atof(input4);
    float hfrac = atof(input5);
    float lam   = atof(input6);

    int K  = 2*k + 2;
    uint Nmn = N-n;
    int mappingLen, imageLen, i = 0;

    // getting the lengths of mappingindices and images
    while (getc(fp) != EOF) { mappingLen++; }
    while (getc(fpim) != EOF) { imageLen++; }

    // rewinding the pointer to extract the data
    rewind(fpim);

    // creating each array
    char mappings[mappingLen];

    // scanning mappingindices
    fscanf(fpim, " %[^\n] ", mappings);

    // allocating host memory for mappingindices and pixels
    int* h_mappingindices = (int*) calloc(N,sizeof(int));
    float* h_Y = (float*) calloc(N*m,sizeof(float));


    char buff[1000];
    for (int i = 0; i < N*m; i++) {
      readNum(fpim, buff);
      h_Y[i] = atof(buff);
    }

    // converting mappingindices from char* to int*
    char delim[] = ",";
    char *mapPtr = strtok(mappings, delim);


    // inserting data to mappingindices
    while(mapPtr != NULL) {
        h_mappingindices[i] = atoi(mapPtr);
        i++;
        mapPtr = strtok(NULL, delim);
    }

    // closing file with data
    fclose(fp);

    // allocate device memory
    uint map_size = N*sizeof(int);
    uint sam_size = N*m*sizeof(float);
    int* d_mappingindices;
    float* d_Y;
    hipMalloc((void**) &d_mappingindices, map_size);
    hipMalloc((void**) &d_Y, sam_size);


    // copy host memory to device
    hipMemcpy(d_mappingindices, h_mappingindices, map_size, hipMemcpyHostToDevice);
    hipMemcpy(d_Y, h_Y, sam_size, hipMemcpyHostToDevice);

    uint X_size     = K*N*sizeof(float);
    uint Y_size     = N*m*sizeof(float);
    uint Nss_size   = N*m*sizeof(uint);
    uint nss_size   = m*sizeof(uint);
    uint hs_size    = m*sizeof(int);
    uint sigmas_size= m*sizeof(float);
    uint I_size     = N*m*sizeof(int);
    uint Xsqr_size  = K*K*m*sizeof(float);
    uint B0_size    = K*m*sizeof(float);
    uint MO_size    = m*sizeof(float);
    uint MOp_size   = Nmn*sizeof(float);
    uint MOpp_size  = Nmn*sizeof(float);
    uint means_size = m*sizeof(float);
    uint breaks_size= m*sizeof(int);
    uint bound_size = (N-n)*sizeof(float);
    // Tile size for register tiling
    const uint R = 30;

    // allocate host memory for X
    float* h_X      = (float*) calloc(N*K,sizeof(float));
    float* h_XT     = (float*) calloc(K*N,sizeof(float));
    float* h_Xsqr   = (float*) calloc(K*K*m,sizeof(float));
    float* h_B0     = (float*) calloc(K*m,sizeof(float));
    float* h_B      = (float*) calloc(K*m,sizeof(float));
    float* h_yhat   = (float*) calloc(N*m,sizeof(float));
    float* h_Xinv   = (float*) calloc(K*K*m,sizeof(float));
    float* h_yerrs  = (float*) calloc(m*N,sizeof(float));
    uint * h_Nss    = (uint *) calloc(m*N,sizeof(uint));
    uint * h_nss    = (uint *) calloc(m,sizeof(uint));
    int  * h_hs     = (int  *) calloc(m,sizeof(int));
    float* h_sigmas = (float*) calloc(m,sizeof(float));
    int  * h_indss  = (int  *) calloc(m*N,sizeof(int));
    float* h_MOfsts = (float*) calloc(m,sizeof(float));
    float* h_bounds = (float*) calloc(N-n,sizeof(float));
    int*   h_breaks = (int*) calloc(m,sizeof(int));
    float* h_means = (float*) calloc(m,sizeof(float));

    // allocate device memory for X, XT and Xsqr
    float *d_X, *d_XT, *d_Xsqr, *d_Xinv, *d_YT, *d_B0, *d_B, *d_yhat;
    float *d_yerall, *d_yerrs, *d_MOfsts, *d_sigmas, *d_MOp, *d_means, *d_MOpp, *d_bounds;
    uint  *d_Nss, *d_nss;
    int   *d_indss, *d_hs, *d_breaks;
    hipMalloc((void**) &d_X, X_size);
    hipMalloc((void**) &d_XT, X_size);
    hipMalloc((void**) &d_YT, Y_size);
    hipMalloc((void**) &d_Xsqr, Xsqr_size);
    hipMalloc((void**) &d_Xinv, Xsqr_size);
    hipMalloc((void**) &d_B0, B0_size);
    hipMalloc((void**) &d_B, B0_size);
    hipMalloc((void**) &d_yhat, Y_size);
    hipMalloc((void**) &d_yerall, Y_size);
    hipMalloc((void**) &d_yerrs, Y_size);
    hipMalloc((void**) &d_Nss, Nss_size);
    hipMalloc((void**) &d_nss, nss_size);
    hipMalloc((void**) &d_hs, hs_size);
    hipMalloc((void**) &d_sigmas, sigmas_size);
    hipMalloc((void**) &d_indss, I_size);
    hipMalloc((void**) &d_hs, MO_size);
    hipMalloc((void**) &d_MOfsts, MO_size);
    hipMalloc((void**) &d_MOp, MOp_size);
    hipMalloc((void**) &d_MOpp, MOpp_size);
    hipMalloc((void**) &d_breaks, breaks_size);
    hipMalloc((void**) &d_means, means_size);
    hipMalloc((void**) &d_bounds, bound_size);


    /////////////////////////////////////////////////////////////////////////
    //// KERNEL 1
    /////////////////////////////////////////////////////////////////////////
    {
        dim3 block(256, 1, 1);
        dim3 grid ((N*K+256-1)/256, 1, 1);

        unsigned long int elapsed;
        struct timeval t_start, t_end, t_diff;
        gettimeofday(&t_start, NULL);

        // GPU call to kernel 1
        //   ker1 <<< grid, block >>>(N, K, freq, d_mappingindices, d_X, d_XT);
        // hipDeviceSynchronize();
        mkX(N, K, freq, h_mappingindices, h_X);
        transpose(N, K, h_X, h_XT);

        gettimeofday(&t_end, NULL);
        timeval_subtract(&t_diff, &t_end, &t_start);
        elapsed = (t_diff.tv_sec*1e6+t_diff.tv_usec);

        // check for cuda errors
        gpuAssert( hipPeekAtLastError() );

        // copy result from device to host
        // hipMemcpy(h_X, d_X, X_size, hipMemcpyDeviceToHost);
        // hipMemcpy(h_XT, d_XT, X_size, hipMemcpyDeviceToHost);


    //--------------------------------------------------------------------------
    // X validation with the sequential version
    //--------------------------------------------------------------------------
        // copy host memory to device
        hipMemcpy(d_X, h_X, X_size, hipMemcpyHostToDevice);
        hipMemcpy(d_XT, h_XT, X_size, hipMemcpyHostToDevice);
    //--------------------------------------------------------------------------

        // add to validation
        printX(fpV, h_X, K, N);

        printf("GPU sequential Kernel 1 runs in: %lu microsecs\n", elapsed);
   }


    /////////////////////////////////////////////////////////////////////////
    //// KERNEL 2
    /////////////////////////////////////////////////////////////////////////
    {
        dim3 block(K, K, 1);
        dim3 grid ((m+R-1)/R, 1, 1);

        unsigned long int elapsed;
        struct timeval t_start, t_end, t_diff;
        gettimeofday(&t_start, NULL);

        //  transpose Y for kernel 2 optimization
        //  GPU call to kernel 2
        // transposeTiled(d_Y, d_YT, m, N, 32);
        // ker2 <<< grid, block >>> (n, N, m, d_X, d_XT, d_YT, d_Xsqr, K);
        // ker2naive <<< grid, block >>> (n, N, m, d_X, d_XT, d_YT, d_Xsqr, K);
        // ker2tiled(n, N, m, h_X, h_XT, h_Y, h_Xsqr, K, R);
        mkXsqrG(n, N, m, h_X, h_XT, h_Y, h_Xsqr, K);


        hipDeviceSynchronize();

        gettimeofday(&t_end, NULL);
        timeval_subtract(&t_diff, &t_end, &t_start);
        elapsed = (t_diff.tv_sec*1e6+t_diff.tv_usec);

        // check for cuda errors
        gpuAssert( hipPeekAtLastError() );

        // copy result from device to host
        // hipMemcpy(h_Xsqr, d_Xsqr, Xsqr_size, hipMemcpyDeviceToHost);

        // validation
        hipMemcpy(d_Xsqr, h_Xsqr, X_size, hipMemcpyHostToDevice);
        printM(fpV, h_Xsqr, m, K);

        printf("SEQ Naive Kernel 2 runs in: %lu microsecs\n", elapsed);
    }

    /////////////////////////////////////////////////////////////////////////
    //// KERNEL 3
    /////////////////////////////////////////////////////////////////////////
    {
        dim3 block(2*K, K, 1);
        dim3 grid (m, 1, 1);

        unsigned long int elapsed;
        struct timeval t_start, t_end, t_diff;
        gettimeofday(&t_start, NULL);

        // GPU call to kernel 3
        //   ker3<<< grid, block, 4*K*K*sizeof(float) >>>(m, K, d_Xsqr, d_Xinv);
        // hipDeviceSynchronize();
        //------------------------------------------------------------------------------
        // X validation with the sequential version
        //------------------------------------------------------------------------------
            // mkXsqrInv(m, h_seq_Xsqr, h_seq_XInv, K);
            gaussJordanG(m, K, h_Xsqr, h_Xinv);
            hipMemcpy(d_Xinv, h_Xinv, Xsqr_size, hipMemcpyHostToDevice);
        //------------------------------------------------------------------------------

        gettimeofday(&t_end, NULL);
        timeval_subtract(&t_diff, &t_end, &t_start);
        elapsed = (t_diff.tv_sec*1e6+t_diff.tv_usec);

        // check for cuda errors
        // gpuAssert( hipPeekAtLastError() );

        // copy result from device to host
        hipMemcpy(h_Xinv, d_Xinv, Xsqr_size, hipMemcpyDeviceToHost);

        // validation
        printM(fpV, h_Xinv, m, K);


        printf("GPU Sequential Kernel 3 runs in: %lu microsecs\n", elapsed);
    }


    /////////////////////////////////////////////////////////////////////////
    //// KERNEL 4
    /////////////////////////////////////////////////////////////////////////
    {
        dim3 block(K, K, 1);
        dim3 grid ((m+K-1)/K, 1, 1);

        unsigned long int elapsed;
        struct timeval t_start, t_end, t_diff;
        gettimeofday(&t_start, NULL);

        // GPU call to kernel 4
        ker4 <<< grid, block >>> (m, n, N, d_X, K, d_Y, d_B0);
        hipDeviceSynchronize();

        gettimeofday(&t_end, NULL);
        timeval_subtract(&t_diff, &t_end, &t_start);
        elapsed = (t_diff.tv_sec*1e6+t_diff.tv_usec);

        // check for cuda errors
        gpuAssert( hipPeekAtLastError() );

        // copy result from device to host
        hipMemcpy(h_B0, d_B0, B0_size, hipMemcpyDeviceToHost);

        // add to validation
        printVf(fpV, h_B0, m, K);

        printf("GPU Naive Kernel 4 runs in: %lu microsecs\n", elapsed);
    }

    /////////////////////////////////////////////////////////////////////////
    //// KERNEL 5
    /////////////////////////////////////////////////////////////////////////
    {
        dim3 block(K, K, 1);
        dim3 grid ((m+K-1)/K, 1, 1);

        unsigned long int elapsed;
        struct timeval t_start, t_end, t_diff;
        gettimeofday(&t_start, NULL);

        // GPU call to kernel 5
        // ker5 <<< grid, block >>> (m, d_Xinv, K, d_B0, d_B);
        ker5OP <<< grid, block >>> (m, d_Xinv, K, d_B0, d_B);
        hipDeviceSynchronize();

        gettimeofday(&t_end, NULL);
        timeval_subtract(&t_diff, &t_end, &t_start);
        elapsed = (t_diff.tv_sec*1e6+t_diff.tv_usec);

        // check for cuda errors
        gpuAssert( hipPeekAtLastError() );

        // copy result from device to host
        hipMemcpy(h_B, d_B, B0_size, hipMemcpyDeviceToHost);

        // validation
        printVf(fpV, h_B, m, K);

        printf("GPU Naive Kernel 5 runs in: %lu microsecs\n", elapsed);
    }

    /////////////////////////////////////////////////////////////////////////
    //// KERNEL 6
    /////////////////////////////////////////////////////////////////////////
    {
        int  dimx = ceil( ((float) N)/ K);
        int  dimy = ceil( ((float) m)/ K);
        dim3 block(K, K, 1);
        dim3 grid (dimx, dimy, 1);

        unsigned long int elapsed;
        struct timeval t_start, t_end, t_diff;
        gettimeofday(&t_start, NULL);

        // GPU call to kernel 6
        ker6 <<< grid, block >>> (m, N, d_XT, d_B, K, d_yhat);
        hipDeviceSynchronize();

        gettimeofday(&t_end, NULL);
        timeval_subtract(&t_diff, &t_end, &t_start);
        elapsed = (t_diff.tv_sec*1e6+t_diff.tv_usec);

        // check for cuda errors
        gpuAssert( hipPeekAtLastError() );

        // copy result from device to host
        hipMemcpy(h_yhat, d_yhat, Y_size, hipMemcpyDeviceToHost);

        // validation
        printVf(fpV, h_yhat, m, N);

        printf("GPU Naive Kernel 6 runs in: %lu microsecs\n", elapsed);
    }

    /////////////////////////////////////////////////////////////////////////
    //// KERNEL 7
    /////////////////////////////////////////////////////////////////////////
    {
        dim3 block(N, 1, 1);
        dim3 grid (m, 1, 1);

        unsigned long int elapsed;
        struct timeval t_start, t_end, t_diff;
        gettimeofday(&t_start, NULL);

        // GPU call to kernel 7
        ker7 <<< grid, block, 2*N*sizeof(float) >>> (m, N, d_yhat, d_yerall, d_Nss, d_yerrs, d_Y, d_indss);
        hipDeviceSynchronize();

        gettimeofday(&t_end, NULL);
        timeval_subtract(&t_diff, &t_end, &t_start);
        elapsed = (t_diff.tv_sec*1e6+t_diff.tv_usec);

        // check for cuda errors
        gpuAssert( hipPeekAtLastError() );

        // copy result from device to host
        hipMemcpy(h_Nss, d_Nss, Nss_size, hipMemcpyDeviceToHost);
        hipMemcpy(h_yerrs, d_yerrs, Y_size, hipMemcpyDeviceToHost);
        hipMemcpy(h_indss, d_indss, I_size, hipMemcpyDeviceToHost);

        // validation
        printE(fpV, h_Nss, m);
        printVfnan(fpV, h_yerrs, m, N);
        printVi(fpV, h_indss, m, N);

        printf("GPU Optimized Kernel 7 runs in: %lu microsecs\n", elapsed);
    }

    /////////////////////////////////////////////////////////////////////////
    //// KERNEL 8
    /////////////////////////////////////////////////////////////////////////
    {
        dim3 block(n, 1, 1);
        dim3 grid (m, 1, 1);

        unsigned long int elapsed;
        struct timeval t_start, t_end, t_diff;
        gettimeofday(&t_start, NULL);

        // size of the shared memory to be available per block
        int sh_size = n*sizeof(uint) + n*sizeof(float);
        // GPU call to kernel 8
        ker8optim<<< grid, block, sh_size >>>(m, n, N, K, hfrac,
                                    d_yerrs, d_Y,
                                    d_nss, d_hs, d_sigmas);
        hipDeviceSynchronize();

        gettimeofday(&t_end, NULL);
        timeval_subtract(&t_diff, &t_end, &t_start);
        elapsed = (t_diff.tv_sec*1e6+t_diff.tv_usec);

        // check for cuda errors
        gpuAssert( hipPeekAtLastError() );

        // copy result from device to host
        hipMemcpy(h_nss, d_nss, m*sizeof(uint), hipMemcpyDeviceToHost);
        hipMemcpy(h_hs, d_hs, m*sizeof(int), hipMemcpyDeviceToHost);
        hipMemcpy(h_sigmas, d_sigmas, m*sizeof(float), hipMemcpyDeviceToHost);

        // validation
        printE(fpV,  h_nss, m);
        printEi(fpV, h_hs,  m);
        printEf(fpV, h_sigmas,  m);

        printf("GPU Naive Kernel 8 runs in: %lu microsecs\n", elapsed);
    }


    /////////////////////////////////////////////////////////////////////////
    //// KERNEL 9
    /////////////////////////////////////////////////////////////////////////
    {
        dim3 block((n*hfrac), 1, 1);
        dim3 grid (m, 1, 1);

        unsigned long int elapsed;
        struct timeval t_start, t_end, t_diff;
        gettimeofday(&t_start, NULL);

        // GPU call to kernel 9
        ker9 <<< grid, block, (n*hfrac)*sizeof(float) >>> (hfrac, n, m, N, d_hs, d_yerrs, d_nss, d_MOfsts);
        hipDeviceSynchronize();

        gettimeofday(&t_end, NULL);
        timeval_subtract(&t_diff, &t_end, &t_start);
        elapsed = (t_diff.tv_sec*1e6+t_diff.tv_usec);

        // check for cuda errors
        gpuAssert( hipPeekAtLastError() );

        // copy result from device to host
        hipMemcpy(h_MOfsts, d_MOfsts, MO_size, hipMemcpyDeviceToHost);

        // validation
        printEf(fpV, h_MOfsts, m);

        printf("GPU Optimized Kernel 9 runs in: %lu microsecs\n", elapsed);
    }


    /////////////////////////////////////////////////////////////////////////
    //// KERNEL 10
    /////////////////////////////////////////////////////////////////////////
    {
        dim3 block(N-n, 1, 1);
        dim3 grid (m, 1, 1);
        unsigned long int elapsed;
        struct timeval t_start, t_end, t_diff;
        gettimeofday(&t_start, NULL);

        compBound(lam, n, N, Nmn, h_mappingindices, h_bounds);
        hipMemcpy(d_bounds, h_bounds, bound_size, hipMemcpyHostToDevice);

        // GPU call to kernel 10
        ker10 <<< grid, block, (N-n)*sizeof(float) >>> (lam, m, n, N, d_bounds,
                                d_Nss, d_nss, d_sigmas,  d_hs,
                                d_mappingindices, d_MOfsts,
                                d_yerrs, d_indss,  d_MOp,
                                d_means, d_breaks, d_MOpp);
                                hipDeviceSynchronize();

        gettimeofday(&t_end, NULL);
        timeval_subtract(&t_diff, &t_end, &t_start);
        elapsed = (t_diff.tv_sec*1e6+t_diff.tv_usec);

        // check for cuda errors
        gpuAssert( hipPeekAtLastError() );

        // copy result from device to host
        hipMemcpy(h_breaks, d_breaks, breaks_size, hipMemcpyDeviceToHost);
        hipMemcpy(h_means, d_means, means_size, hipMemcpyDeviceToHost);
        printEi(fpV, h_breaks, m);
        printEf(fpV, h_means, m);

        printf("GPU Optimized Kernel 10 runs in: %lu microsecs\n", elapsed);
    }


   fclose(fpV);

   // 7. clean up memory
   free(h_mappingindices);
   free(h_Y);
   free(h_X);
   free(h_XT);
   free(h_Xsqr);
   free(h_Xinv);
   free(h_B0);
   free(h_B);
   free(h_yhat);
   free(h_yerrs);
   free(h_Nss);
   free(h_indss);
   free(h_hs);
   free(h_MOfsts);
   hipFree(d_X);
   hipFree(d_XT);
   hipFree(d_YT);
   hipFree(d_Xsqr);
   hipFree(d_Xinv);
   hipFree(d_B0);
   hipFree(d_B);
   hipFree(d_yhat);
   hipFree(d_yerall);
   hipFree(d_yerrs);
   hipFree(d_Nss);
   hipFree(d_indss);
   hipFree(d_hs);
   hipFree(d_MOfsts);
   hipFree(d_mappingindices);
   hipFree(d_Y);

}


