#include "hip/hip_runtime.h"
#include <sys/time.h>
#include <string.h>
#include <stdlib.h>
#include <stdio.h>
#include <float.h>
#include <math.h>
#include <time.h>

#include "helper.cu.h"
#include "kernels-optim.cu.h"
#include "sequential.cu.h"

#define BLOCK_SIZE 1024//1024 //1024//2048
#define WIDTH_A  1024//1024 //1024//2048
#define HEIGHT_A 1//2048//2048//2048
#define WIDTH_B  1024//4096//2048
#define TILE_HEIGHT 1
#define TILE_WIDTH 1024

#define F32_MIN -FLT_MAX
#define I32_MIN -2147483648
typedef unsigned int uint;


/////////////////////////////////////////////////////////////////////////
/////////////////////////////////////////////////////////////////////////
//// Helpers
/////////////////////////////////////////////////////////////////////////
/////////////////////////////////////////////////////////////////////////

int timeval_subtract(struct timeval *result, struct timeval *t2, struct timeval *t1) {
    unsigned int resolution=1000000;
    long int diff = (t2->tv_usec + resolution * t2->tv_sec) - (t1->tv_usec + resolution * t1->tv_sec);
    result->tv_sec = diff / resolution;
    result->tv_usec = diff % resolution;
    return (diff<0);
}


void randomInit(float* data, int size) {
   for (int i = 0; i < size; ++i)
   data[i] = rand() / (float)RAND_MAX;
}


template<class T>
void matMult(T* A, T* B, T* C, int colsA, int rowsA, int colsB) {
  for(int i = 0; i < rowsA; i++) {
    for(int j = 0; j < colsB; j++) {
      float sum = 0.0;
      for(int k = 0; k < colsA; k++) {
        sum += A[i*colsA + k] * B[k * colsB + j];
      }
      C[i * colsB + j] = sum;
    }
  }
}

template<class T>
bool validate(float* A,float* B, unsigned int sizeAB){
    for(int i = 0; i < sizeAB; i++)
      if (fabs(A[i] - B[i]) > 0.0005){
        printf("INVALID RESULT %d %f %f\n", i, A[i], B[i]);
        return false;
      }
    printf("VALID RESULT!\n");
    return true;
}

int gpuAssert(hipError_t code) {
    if(code != hipSuccess) {
      printf("GPU Error: %s\n", hipGetErrorString(code));
      return -1;
    }
    return 0;
}


///////////////////////////////////////////////////////////////////////////////
///////////////////////////////////////////////////////////////////////////////
//// PROGRAM MAIN
///////////////////////////////////////////////////////////////////////////////
///////////////////////////////////////////////////////////////////////////////

int main(int argc, char const *argv[]) {
   if (argc != 2) {
      printf("Please include the name of the dataset.\n");
         return -1;
   }

///////////////////////////////////////////////////////////////////////////////
//// PARSING
///////////////////////////////////////////////////////////////////////////////
   FILE *fp, *fpim;

   if (argv[1][0] == 's') {
      fp   = fopen("../data/saharaC.in", "r");
      fpim = fopen("../data/saharaCimages.in", "r");
   } else {
      fp   = fopen("../data/peruC.in", "r");
      fpim = fopen("../data/peruCimages.in", "r");
   }

   if (fp == NULL || fpim == NULL) {
      printf("Files not read.\n");
      return -1;
   }

   char input1[10], input2[10], input3[30], input4[30];
   char input5[30], input6[30], input7[50], input8[30];
   fscanf(fp, " %[^\n]  %[^\n]  %[^\n]  %[^\n] ", input1,input2,input3,input4);
   fscanf(fp, " %[^\n]  %[^\n]  %[^\n]  %[^\n] ", input5,input6,input7,input8);

   int  k    = atoi(input2);
   uint n    = (uint)atoi(input3);
   uint N    = (uint)atoi(input8);
   uint mIRL = (uint)atoi(input7);
   int trend = atoi(input1);
   float freq  = atof(input4);
   float hfrac = atof(input5);
   float lam   = atof(input6);
   uint m = 2;
   int K  = 2*k + 2;

   int mappingLen, imageLen, i = 0;

   // getting the lengths of mappingindices and images
   while (getc(fp) != EOF) { mappingLen++; }
   while (getc(fpim) != EOF) { imageLen++; }

   // rewinding the pointer to extract the data
   rewind(fpim);

   // extracting each array
   char mappings[mappingLen], pixels[(imageLen-mappingLen)];
   fscanf(fpim, " %[^\n]  %[^\n] ", mappings, pixels);

   // converting mappingindices from char* to int*
   char delim[] = ",";
   char *mapPtr = strtok(mappings, delim);

   // allocating host memory for mappingindices and pixels
   int* h_mappingindices = (int*) calloc(N,sizeof(int));
   float* h_sample = (float*) calloc(N*m,sizeof(float));

   // inserting data to mappingindices
   while(mapPtr != NULL) {
      h_mappingindices[i] = atoi(mapPtr);
      i++;
      mapPtr = strtok(NULL, delim);
   }

   // converting samples from char* to float*
   char *pixelsPtr = strtok(pixels, delim);
   i = 0;

   // inserting data to sample
   while(pixelsPtr != NULL) {
      h_sample[i] = atof(pixelsPtr);
      i++;
      pixelsPtr = strtok(NULL, delim);
   }

   // closing file with data
   fclose(fp);

   // opening file for validation of results
   FILE* fpV = fopen("../data/val.data","a+");


   // allocate device memory
   uint map_size = N*sizeof(int);
   uint sam_size = N*m*sizeof(float);
   int* d_mappingindices;
   float* d_sample;
   hipMalloc((void**) &d_mappingindices, map_size);
   hipMalloc((void**) &d_sample, sam_size);

   // copy host memory to device
   hipMemcpy(d_mappingindices, h_mappingindices, map_size, hipMemcpyHostToDevice);
   hipMemcpy(d_sample, h_sample, sam_size, hipMemcpyHostToDevice);

   uint X_size     = K*N*sizeof(float);
   uint Xsqr_size  = K*K*m*sizeof(float);
   uint Xinv_size  = K*K*m*sizeof(float);
   uint B0_size    = K*m*sizeof(float);

   // allocate host memory for X
   float* h_X      = (float*) calloc(N*K,sizeof(float));
   float* h_XT     = (float*) calloc(K*N,sizeof(float));
   float* h_Xsqr   = (float*) calloc(K*K*m,sizeof(float));
   float* h_Xinv   = (float*) calloc(K*K*m,sizeof(float));
   float* h_B0     = (float*) calloc(K*m,sizeof(float));

   // allocate device memory for X, XT and Xsqr
   float *d_X, *d_XT, *d_Xsqr, *d_Xinv, *d_B0;
   hipMalloc((void**) &d_X, X_size);
   hipMalloc((void**) &d_XT, X_size);
   hipMalloc((void**) &d_Xsqr, Xsqr_size);
   hipMalloc((void**) &d_Xinv, Xinv_size);
   hipMalloc((void**) &d_B0, B0_size);


   /////////////////////////////////////////////////////////////////////////
   //// KERNEL 1
   /////////////////////////////////////////////////////////////////////////
   {
      dim3 block(1024, 1, 1);
      dim3 grid (1024, 1, 1);

      unsigned long int elapsed;
      struct timeval t_start, t_end, t_diff;
      gettimeofday(&t_start, NULL);

      // GPU call to kernel 1
      ker1 <<< grid, block >>>(N, K, freq, d_mappingindices, d_X, d_XT);
      hipDeviceSynchronize();

      gettimeofday(&t_end, NULL);
      timeval_subtract(&t_diff, &t_end, &t_start);
      elapsed = (t_diff.tv_sec*1e6+t_diff.tv_usec);

      // check for cuda errors
      gpuAssert( hipPeekAtLastError() );

      // copy result from device to host
      hipMemcpy(h_X, d_X, X_size, hipMemcpyDeviceToHost);
      hipMemcpy(h_XT, d_XT, X_size, hipMemcpyDeviceToHost);

      // add to validation
      printX(fpV, h_X, K, N);

      printf("GPU Optimized Kernel 1 runs in: %lu microsecs\n", elapsed);
      float microsecPerMatrixMul = elapsed;
      double flopsPerMatrixMul = 2.0 * HEIGHT_A * WIDTH_B * WIDTH_A;
      // double gigaFlops = (flopsPerMatrixMul * 1.0e-9f) / (microsecPerMatrixMul / (1000.0f * 1000.0f));
      // printf( "GPU Optimized Kernel 1 Performance= %.2f GFlop/s, Time= %.3f microsec %d %d\n", gigaFlops, microsecPerMatrixMul, grid.x, grid.y);
   }


   /////////////////////////////////////////////////////////////////////////
   //// KERNEL 2
   /////////////////////////////////////////////////////////////////////////
   {
      dim3 block(K, K, 1);
      dim3 grid (m, 1, 1);

      unsigned long int elapsed;
      struct timeval t_start, t_end, t_diff;
      gettimeofday(&t_start, NULL);

      // GPU call to kernel 2
      ker2 <<< grid, block >>> (n, N, m, d_X, d_XT, d_sample, d_Xsqr, K);
      hipDeviceSynchronize();

      gettimeofday(&t_end, NULL);
      timeval_subtract(&t_diff, &t_end, &t_start);
      elapsed = (t_diff.tv_sec*1e6+t_diff.tv_usec);

      // check for cuda errors
      gpuAssert( hipPeekAtLastError() );

      // copy result from device to host
      hipMemcpy(h_Xsqr, d_Xsqr, Xsqr_size, hipMemcpyDeviceToHost);

      // validation
      printM(fpV, h_Xsqr, m, K);


      printf("GPU Optimized Kernel 2 runs in: %lu microsecs\n", elapsed);
      float microsecPerMatrixMul = elapsed;
      double flopsPerMatrixMul = 2.0 * HEIGHT_A * WIDTH_B * WIDTH_A;
      // double gigaFlops = (flopsPerMatrixMul * 1.0e-9f) / (microsecPerMatrixMul / (1000.0f * 1000.0f));
      // printf( "GPU Optimized Kernel 2 Performance= %.2f GFlop/s, Time= %.3f microsec %d %d\n", gigaFlops, microsecPerMatrixMul, grid.x, grid.y);
   }

   /////////////////////////////////////////////////////////////////////////
   //// KERNEL 3
   /////////////////////////////////////////////////////////////////////////
   {
    dim3 block(K, K, 1);
    dim3 grid (n, 1, 1);

    unsigned long int elapsed;
    struct timeval t_start, t_end, t_diff;
    gettimeofday(&t_start, NULL);

    // GPU call to kernel 3
    ker3<<< grid, block >>>(m, K, d_Xsqr, d_Xinv);
    // hipDeviceSynchronize();

    gettimeofday(&t_end, NULL);
    timeval_subtract(&t_diff, &t_end, &t_start);
    elapsed = (t_diff.tv_sec*1e6+t_diff.tv_usec);

    // check for cuda errors
    gpuAssert( hipPeekAtLastError() );

    // copy result from device to host
    hipMemcpy(h_Xinv, d_Xinv, Xinv_size, hipMemcpyDeviceToHost);
    printM(fpV, h_Xinv, m, K);
    // printM(fpV, h_Xsqr, m, K);

    printf("GPU Optimized Kernel 3 runs in: %lu microsecs\n", elapsed);
    float microsecPerMatrixMul = elapsed;
    double flopsPerMatrixMul = 2.0 * HEIGHT_A * WIDTH_B * WIDTH_A;
    double gigaFlops = (flopsPerMatrixMul * 1.0e-9f) / (microsecPerMatrixMul / (1000.0f * 1000.0f));
    // printf( "GPU Optimized Kernel 3 Performance= %.2f GFlop/s, Time= %.3f microsec %d %d\n", gigaFlops, microsecPerMatrixMul, grid.x, grid.y);
   }


   /////////////////////////////////////////////////////////////////////////
   //// KERNEL 4
   /////////////////////////////////////////////////////////////////////////
   {
      dim3 block(K, K, 1);
      dim3 grid (m, 1, 1);

      unsigned long int elapsed;
      struct timeval t_start, t_end, t_diff;
      gettimeofday(&t_start, NULL);

      // GPU call to kernel 4
      ker4 <<< grid, block >>> (m, n, N, d_X, K, d_sample, d_B0);
      hipDeviceSynchronize();

      gettimeofday(&t_end, NULL);
      timeval_subtract(&t_diff, &t_end, &t_start);
      elapsed = (t_diff.tv_sec*1e6+t_diff.tv_usec);

      // check for cuda errors
      gpuAssert( hipPeekAtLastError() );

      // copy result from device to host
      hipMemcpy(h_B0, d_B0, B0_size, hipMemcpyDeviceToHost);

      // add to validation
    //   printVf(fpV, h_B0, m, K);

      printf("GPU Optimized Kernel 4 runs in: %lu microsecs\n", elapsed);
      float microsecPerMatrixMul = elapsed;
      double flopsPerMatrixMul = 2.0 * HEIGHT_A * WIDTH_B * WIDTH_A;
      double gigaFlops = (flopsPerMatrixMul * 1.0e-9f) / (microsecPerMatrixMul / (1000.0f * 1000.0f));
      // printf( "GPU Optimized Kernel 4 Performance= %.2f GFlop/s, Time= %.3f microsec %d %d\n", gigaFlops, microsecPerMatrixMul, grid.x, grid.y);
   }

   /////////////////////////////////////////////////////////////////////////
   //// KERNEL 5
   /////////////////////////////////////////////////////////////////////////
   {
      int  dimx = ceil( ((float) WIDTH_B)/TILE_HEIGHT );
      int  dimy = ceil( ((float)HEIGHT_A)/TILE_WIDTH );
      dim3 block(TILE_WIDTH, TILE_HEIGHT, 1);
      dim3 grid (dimx, dimy, 1);

      unsigned long int elapsed;
      struct timeval t_start, t_end, t_diff;
      gettimeofday(&t_start, NULL);

      // GPU call to kernel 5
      // ker5 <<< grid, block >>> ();
      // hipDeviceSynchronize();

      gettimeofday(&t_end, NULL);
      timeval_subtract(&t_diff, &t_end, &t_start);
      elapsed = (t_diff.tv_sec*1e6+t_diff.tv_usec);

      // check for cuda errors
      gpuAssert( hipPeekAtLastError() );

      // copy result from device to host
      // hipMemcpy(h_X, d_X, X_size, hipMemcpyDeviceToHost);


      printf("GPU Optimized Kernel 5 runs in: %lu microsecs\n", elapsed);
      float microsecPerMatrixMul = elapsed;
      double flopsPerMatrixMul = 2.0 * HEIGHT_A * WIDTH_B * WIDTH_A;
      double gigaFlops = (flopsPerMatrixMul * 1.0e-9f) / (microsecPerMatrixMul / (1000.0f * 1000.0f));
      printf( "GPU Optimized Kernel 5 Performance= %.2f GFlop/s, Time= %.3f microsec %d %d\n", gigaFlops, microsecPerMatrixMul, grid.x, grid.y);
   }


   /////////////////////////////////////////////////////////////////////////
   //// KERNEL 6
   /////////////////////////////////////////////////////////////////////////
   {
      int  dimx = ceil( ((float) WIDTH_B)/TILE_HEIGHT );
      int  dimy = ceil( ((float)HEIGHT_A)/TILE_WIDTH );
      dim3 block(TILE_WIDTH, TILE_HEIGHT, 1);
      dim3 grid (dimx, dimy, 1);

      unsigned long int elapsed;
      struct timeval t_start, t_end, t_diff;
      gettimeofday(&t_start, NULL);

      // GPU call to kernel 6
      // ker6 <<< grid, block >>> ();
      // hipDeviceSynchronize();

      gettimeofday(&t_end, NULL);
      timeval_subtract(&t_diff, &t_end, &t_start);
      elapsed = (t_diff.tv_sec*1e6+t_diff.tv_usec);

      // check for cuda errors
      gpuAssert( hipPeekAtLastError() );

      // copy result from device to host
      // hipMemcpy(h_X, d_X, X_size, hipMemcpyDeviceToHost);

      printf("GPU Optimized Kernel 6 runs in: %lu microsecs\n", elapsed);
      float microsecPerMatrixMul = elapsed;
      double flopsPerMatrixMul = 2.0 * HEIGHT_A * WIDTH_B * WIDTH_A;
      double gigaFlops = (flopsPerMatrixMul * 1.0e-9f) / (microsecPerMatrixMul / (1000.0f * 1000.0f));
      printf( "GPU Optimized Kernel 6 Performance= %.2f GFlop/s, Time= %.3f microsec %d %d\n", gigaFlops, microsecPerMatrixMul, grid.x, grid.y);
   }


   /////////////////////////////////////////////////////////////////////////
   //// KERNEL 7
   /////////////////////////////////////////////////////////////////////////
   {
      int  dimx = ceil( ((float) WIDTH_B)/TILE_HEIGHT );
      int  dimy = ceil( ((float)HEIGHT_A)/TILE_WIDTH );
      dim3 block(TILE_WIDTH, TILE_HEIGHT, 1);
      dim3 grid (dimx, dimy, 1);

      unsigned long int elapsed;
      struct timeval t_start, t_end, t_diff;
      gettimeofday(&t_start, NULL);

      // GPU call to kernel 7
      // ker7 <<< grid, block >>> ();
      // hipDeviceSynchronize();

      gettimeofday(&t_end, NULL);
      timeval_subtract(&t_diff, &t_end, &t_start);
      elapsed = (t_diff.tv_sec*1e6+t_diff.tv_usec);

      // check for cuda errors
      gpuAssert( hipPeekAtLastError() );

      // copy result from device to host
      // hipMemcpy(h_X, d_X, X_size, hipMemcpyDeviceToHost);

      printf("GPU Optimized Kernel 7 runs in: %lu microsecs\n", elapsed);
      float microsecPerMatrixMul = elapsed;
      double flopsPerMatrixMul = 2.0 * HEIGHT_A * WIDTH_B * WIDTH_A;
      double gigaFlops = (flopsPerMatrixMul * 1.0e-9f) / (microsecPerMatrixMul / (1000.0f * 1000.0f));
      printf( "GPU Optimized Kernel 7 Performance= %.2f GFlop/s, Time= %.3f microsec %d %d\n", gigaFlops, microsecPerMatrixMul, grid.x, grid.y);
   }


   /////////////////////////////////////////////////////////////////////////
   //// KERNEL 8
   /////////////////////////////////////////////////////////////////////////
   {
      int  dimx = ceil( ((float) WIDTH_B)/TILE_HEIGHT );
      int  dimy = ceil( ((float)HEIGHT_A)/TILE_WIDTH );
      dim3 block(TILE_WIDTH, TILE_HEIGHT, 1);
      dim3 grid (dimx, dimy, 1);

      unsigned long int elapsed;
      struct timeval t_start, t_end, t_diff;
      gettimeofday(&t_start, NULL);

      // GPU call to kernel 8
      // ker8 <<< grid, block >>> ();
      // hipDeviceSynchronize();

      gettimeofday(&t_end, NULL);
      timeval_subtract(&t_diff, &t_end, &t_start);
      elapsed = (t_diff.tv_sec*1e6+t_diff.tv_usec);

      // check for cuda errors
      gpuAssert( hipPeekAtLastError() );

      // copy result from device to host
      // hipMemcpy(h_X, d_X, X_size, hipMemcpyDeviceToHost);

      printf("GPU Optimized Kernel 8 runs in: %lu microsecs\n", elapsed);
      float microsecPerMatrixMul = elapsed;
      double flopsPerMatrixMul = 2.0 * HEIGHT_A * WIDTH_B * WIDTH_A;
      double gigaFlops = (flopsPerMatrixMul * 1.0e-9f) / (microsecPerMatrixMul / (1000.0f * 1000.0f));
      printf( "GPU Optimized Kernel 8 Performance= %.2f GFlop/s, Time= %.3f microsec %d %d\n", gigaFlops, microsecPerMatrixMul, grid.x, grid.y);
   }


   /////////////////////////////////////////////////////////////////////////
   //// KERNEL 9
   /////////////////////////////////////////////////////////////////////////
   {
      int  dimx = ceil( ((float) WIDTH_B)/TILE_HEIGHT );
      int  dimy = ceil( ((float)HEIGHT_A)/TILE_WIDTH );
      dim3 block(TILE_WIDTH, TILE_HEIGHT, 1);
      dim3 grid (dimx, dimy, 1);

      unsigned long int elapsed;
      struct timeval t_start, t_end, t_diff;
      gettimeofday(&t_start, NULL);

      // GPU call to kernel 9
      // ker9 <<< grid, block >>> ();
      // hipDeviceSynchronize();

      gettimeofday(&t_end, NULL);
      timeval_subtract(&t_diff, &t_end, &t_start);
      elapsed = (t_diff.tv_sec*1e6+t_diff.tv_usec);

      // check for cuda errors
      gpuAssert( hipPeekAtLastError() );

      // copy result from device to host
      // hipMemcpy(h_X, d_X, X_size, hipMemcpyDeviceToHost);

      printf("GPU Optimized Kernel 9 runs in: %lu microsecs\n", elapsed);
      float microsecPerMatrixMul = elapsed;
      double flopsPerMatrixMul = 2.0 * HEIGHT_A * WIDTH_B * WIDTH_A;
      double gigaFlops = (flopsPerMatrixMul * 1.0e-9f) / (microsecPerMatrixMul / (1000.0f * 1000.0f));
      printf( "GPU Optimized Kernel 9 Performance= %.2f GFlop/s, Time= %.3f microsec %d %d\n", gigaFlops, microsecPerMatrixMul, grid.x, grid.y);
   }


   /////////////////////////////////////////////////////////////////////////
   //// KERNEL 10
   /////////////////////////////////////////////////////////////////////////
   {
      int  dimx = ceil( ((float) WIDTH_B)/TILE_HEIGHT );
      int  dimy = ceil( ((float)HEIGHT_A)/TILE_WIDTH );
      dim3 block(TILE_WIDTH, TILE_HEIGHT, 1);
      dim3 grid (dimx, dimy, 1);

      unsigned long int elapsed;
      struct timeval t_start, t_end, t_diff;
      gettimeofday(&t_start, NULL);

      // GPU call to kernel 10
      // ker10 <<< grid, block >>> ();
      // hipDeviceSynchronize();

      gettimeofday(&t_end, NULL);
      timeval_subtract(&t_diff, &t_end, &t_start);
      elapsed = (t_diff.tv_sec*1e6+t_diff.tv_usec);

      // check for cuda errors
      gpuAssert( hipPeekAtLastError() );

      // copy result from device to host
      // hipMemcpy(h_X, d_X, X_size, hipMemcpyDeviceToHost);

      printf("GPU Optimized Kernel 10 runs in: %lu microsecs\n", elapsed);
      float microsecPerMatrixMul = elapsed;
      double flopsPerMatrixMul = 2.0 * HEIGHT_A * WIDTH_B * WIDTH_A;
      double gigaFlops = (flopsPerMatrixMul * 1.0e-9f) / (microsecPerMatrixMul / (1000.0f * 1000.0f));
      printf( "GPU Optimized Kernel 10 Performance= %.2f GFlop/s, Time= %.3f microsec %d %d\n", gigaFlops, microsecPerMatrixMul, grid.x, grid.y);
   }


   /////////////////////////////////////////////////////////////////////////
   /////////////////////////////////////////////////////////////////////////
   //// VALIDATION
   /////////////////////////////////////////////////////////////////////////
   /////////////////////////////////////////////////////////////////////////

   fclose(fpV);

   // 7. clean up memory
   free(h_mappingindices);
   free(h_sample);
   free(h_X);
   free(h_XT);
   free(h_Xsqr);
   hipFree(d_X);
   hipFree(d_XT);
   hipFree(d_Xsqr);
   hipFree(d_mappingindices);
   hipFree(d_sample);

}


