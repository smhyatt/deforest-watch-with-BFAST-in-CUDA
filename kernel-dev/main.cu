#include "hip/hip_runtime.h"
#include <sys/time.h>
#include <string.h>
#include <stdlib.h>
#include <stdio.h>
#include <float.h>
#include <math.h>
#include <time.h> 

#include "helper.cu.h"
#include "kernels.cu.h"
#include "sequential.cu.h"

#define WIDTH_A  1024//1024 //1024//2048
#define HEIGHT_A 1024//2048//2048//2048
#define WIDTH_B  4096//2048
#define TILE     16
#define PI 3.14159265
#define F32_MIN -FLT_MAX
#define I32_MIN -2147483648
typedef unsigned int uint;


/////////////////////////////////////////////////////////
// Helpers
/////////////////////////////////////////////////////////

int timeval_subtract(struct timeval *result, struct timeval *t2, struct timeval *t1) {
    unsigned int resolution=1000000;
    long int diff = (t2->tv_usec + resolution * t2->tv_sec) - (t1->tv_usec + resolution * t1->tv_sec);
    result->tv_sec = diff / resolution;
    result->tv_usec = diff % resolution;
    return (diff<0);
}


void randomInit(float* data, int size) {
   for (int i = 0; i < size; ++i)
   data[i] = rand() / (float)RAND_MAX;
}


template<class T>
void matMult(T* A, T* B, T* C, int colsA, int rowsA, int colsB) {
  for(int i = 0; i < rowsA; i++) {
    for(int j = 0; j < colsB; j++) {
      float sum = 0.0;
      for(int k = 0; k < colsA; k++) {
        sum += A[i*colsA + k] * B[k * colsB + j];
      }
      C[i * colsB + j] = sum;
    }
  } 
}

template<class T>
bool validate(float* A,float* B, unsigned int sizeAB){
    for(int i = 0; i < sizeAB; i++)
      if (fabs(A[i] - B[i]) > 0.0005){
        printf("INVALID RESULT %d %f %f\n", i, A[i], B[i]);
        return false;
      }
    printf("VALID RESULT!\n");
    return true;
}

/////////////////////////////////////////////////////////
// Program main
/////////////////////////////////////////////////////////
 
int main(int argc, char const *argv[]) {
   if (argc != 2) {
      printf("Please include the name of the dataset.\n");
         return -1;
   }

// *****************************************************************************
// Parsing
// *****************************************************************************
   
   FILE *fp, *fpim;
   
   if (argv[1][0] == 's') {
      fp   = fopen("data/saharaC.in", "r");
      fpim = fopen("data/saharaCimages.in", "r");
   } else {
      fp   = fopen("data/peruC.in", "r");
      fpim = fopen("data/peruCimages.in", "r");
   }

   char input1[10], input2[10], input3[30], input4[30];
   char input5[30], input6[30], input7[50], input8[30];
   fscanf(fp, " %[^\n]  %[^\n]  %[^\n]  %[^\n] ", input1,input2,input3,input4);
   fscanf(fp, " %[^\n]  %[^\n]  %[^\n]  %[^\n] ", input5,input6,input7,input8);
   
   int  k    = atoi(input2); 
   uint n    = (uint)atoi(input3);
   uint N    = (uint)atoi(input8);
   uint mIRL = (uint)atoi(input7);
   int trend = atoi(input1); 
   float freq  = atof(input4);
   float hfrac = atof(input5);
   float lam   = atof(input6);
   uint m = 2;
   int K  = 2*k + 2;

   int mappingLen, imageLen, i = 0;
   char c;

   // getting the lengths of mappingindices and images
   while ((c = getc(fp)) != EOF) { mappingLen++; }
   while ((c = getc(fpim)) != EOF) { imageLen++; }

   // rewinding the pointer to extract the data
   rewind(fpim);

   // extracting each array
   char mappings[mappingLen], pixels[(imageLen-mappingLen)];
   fscanf(fpim, " %[^\n]  %[^\n] ", mappings, pixels);

   // converting mappingindices from char* to int*
   char delim[] = ",";
   char *mapPtr = strtok(mappings, delim);
 
   // allocating host memory for mappingindices and pixels
   int* h_mappingindices = calloc(N,sizeof(int));
   float* h_sample = calloc(N*m,sizeof(float));

   // inserting data to mappingindices
   while(mapPtr != NULL) {
      h_mappingindices[i] = atoi(mapPtr);
      i++;
      mapPtr = strtok(NULL, delim);
   }

   // converting samples from char* to float*
   char *pixelsPtr = strtok(pixels, delim);
   i = 0;

   // inserting data to sample
   while(pixelsPtr != NULL) {
      h_sample[i] = atof(pixelsPtr);
      i++;
      pixelsPtr = strtok(NULL, delim);
   }

   fclose(fp);

    
   // allocate device memory
   uint map_size = N*m*sizeof(float);
   uint sam_size = N*sizeof(float);
   float* d_mappingindices;
   float* d_sample;
   hipMalloc((void**) &d_mappingindices, map_size);
   hipMalloc((void**) &d_sample, sam_size);
 
   // copy host memory to device
   hipMemcpy(d_mappingindices, h_mappingindices, map_size, hipMemcpyHostToDevice);
   hipMemcpy(d_sample, h_sample, sam_size, hipMemcpyHostToDevice);
 
   // allocate host memory for X
   uint X_size    = K*N*sizeof(float);
   float* h_X     = calloc(K*N,sizeof(float));
   float* h_seq_X = calloc(K*N,sizeof(float));
 
   // allocate device memory for X
   float *d_X;
   hipMalloc((void**) &d_X, X_size);

 
   // compute sequential creation of X and XT
   {
      unsigned long int elapsed;
      struct timeval t_start, t_end, t_diff;
      gettimeofday(&t_start, NULL); 
      
      // calling sequential kernel 1 and transpose from the sequential file 
      ker1(N, K, freq, h_mappingindices, h_X);
      // transpose(N, K, h_X, h_XT);
      // matMult<float>(h_A, h_B, seq_C, WIDTH_A, HEIGHT_A, WIDTH_B);

      gettimeofday(&t_end, NULL);
      timeval_subtract(&t_diff, &t_end, &t_start);
      elapsed = (t_diff.tv_sec*1e6+t_diff.tv_usec); 
      printf("Sequential kernel 1 version runs in: %lu microsecs\n", elapsed);
   }

   
   // execute the block+register tiled kernel
   // ToDo: please fill in the implementation below
   //       (for TILE = 16)
   // {
   //    // 1. you would probably want to compute some valid grid and block here
   //    int  dimy = (HEIGHT_A+TILE-1)/TILE; 
   //    int  dimx = (WIDTH_B+(TILE*TILE)-1)/(TILE*TILE);
   //    dim3 block(TILE, TILE, 1);
   //    dim3 grid (dimx, dimy, 1);

   //    unsigned long int elapsed;
   //    struct timeval t_start, t_end, t_diff;
   //    gettimeofday(&t_start, NULL); 
      
   //    // 2. you would probably want to call here the kernel: 
   //    // matMultRegTiledKer<float,TILE> <<< grid, block >>>(d_A, d_B, d_C, HEIGHT_A, WIDTH_B, WIDTH_A); 
   //    hipDeviceSynchronize();

   //    gettimeofday(&t_end, NULL);
   //    timeval_subtract(&t_diff, &t_end, &t_start);
   //    elapsed = (t_diff.tv_sec*1e6+t_diff.tv_usec); 

   //    // copy result from device to host
   //    hipMemcpy(h_C, d_C, mem_size_C, hipMemcpyDeviceToHost);
   //    // validate
   //    printf("GPU Block+Register Tiled MMM version ... ");
   //    validate<float>(seq_C, h_C, size_C);

   //    printf("GPU Block+Register Tiled MMM version runs in: %lu microsecs\n", elapsed);
   //    float microsecPerMatrixMul = elapsed; 
   //    double flopsPerMatrixMul = 2.0 * HEIGHT_A * WIDTH_B * WIDTH_A; 
   //    double gigaFlops = (flopsPerMatrixMul * 1.0e-9f) / (microsecPerMatrixMul / (1000.0f * 1000.0f)); 
   //    printf( "GPU Block+Register Tiled MMM Performance= %.2f GFlop/s, Time= %.3f microsec %d %d\n", gigaFlops, microsecPerMatrixMul, grid.x, grid.y); 
   // }


   // 7. clean up memory
   free(h_mappingindices);
   free(h_sample);
   free(h_seq_X);
   free(h_X);
   hipFree(d_X);
   hipFree(d_mappingindices);
   hipFree(d_sample);

}



